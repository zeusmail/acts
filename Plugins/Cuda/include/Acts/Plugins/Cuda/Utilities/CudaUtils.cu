// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include <hip/hip_runtime.h>

#include <iostream>

#define ACTS_CUDA_ERROR_CHECK(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
